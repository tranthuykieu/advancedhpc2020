#include "hip/hip_runtime.h"
#include <stdio.h>
#include <include/labwork.h>
#include <hip/hip_runtime_api.h>
#include <omp.h>

#define ACTIVE_THREADS 4

int main(int argc, char **argv) {
    printf("USTH ICT Master 2018, Advanced Programming for HPC.\n");
    if (argc < 2) {
        printf("Usage: labwork <lwNum> <inputImage>\n");
        printf("   lwNum        labwork number\n");
        printf("   inputImage   the input file name, in JPEG format\n");
        return 0;
    }

    int lwNum = atoi(argv[1]);
    std::string inputFilename;

    // pre-initialize CUDA to avoid incorrect profiling
    printf("Warming up...\n");
    char *temp;
    hipMalloc(&temp, 1024);

    Labwork labwork;
    if (lwNum != 2 ) {
        inputFilename = std::string(argv[2]);
        labwork.loadInputImage(inputFilename);
    }

    printf("Starting labwork %d\n", lwNum);
    Timer timer;
    timer.start();
    switch (lwNum) {
        case 1:
            labwork.labwork1_CPU();
            labwork.saveOutputImage("labwork2-cpu-out.jpg");
            printf("labwork 1 CPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            timer.start();
            labwork.labwork1_OpenMP();
            printf("labwork 1 CPU openmp ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork2-openmp-out.jpg");
            break;
        case 2:
            labwork.labwork2_GPU();
            break;
        case 3:
            labwork.labwork3_GPU();
            labwork.saveOutputImage("labwork3-gpu-out.jpg");
            break;
        case 4:
            labwork.labwork4_GPU();
            labwork.saveOutputImage("labwork4-gpu-out.jpg");
            break;
        case 5:
            labwork.labwork5_CPU();
            labwork.saveOutputImage("labwork5-cpu-out.jpg");
            labwork.labwork5_GPU(FALSE);
            labwork.saveOutputImage("labwork5-gpu-out.jpg");
            break;
        case 6:
            labwork.labwork6_GPU();
            labwork.saveOutputImage("labwork6-gpu-out.jpg");
            break;
        case 7:
            labwork.labwork7_GPU();
            printf("[ALGO ONLY] labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork7-gpu-out.jpg");
            break;
        case 8:
            labwork.labwork8_GPU();
            printf("[ALGO ONLY] labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork8-gpu-out.jpg");
            break;
        case 9:
            labwork.labwork9_GPU();
            printf("[ALGO ONLY] labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork9-gpu-out.jpg");
            break;
        case 10:
            labwork.labwork10_GPU();
            printf("[ALGO ONLY] labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork10-gpu-out.jpg");
            break;
    }
    printf("labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
}

void Labwork::loadInputImage(std::string inputFileName) {
    inputImage = jpegLoader.load(inputFileName);
}

void Labwork::saveOutputImage(std::string outputFileName) {
    jpegLoader.save(outputFileName, outputImage, inputImage->width, inputImage->height, 90);
}

void Labwork::labwork1_CPU() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    for (int j = 0; j < 100; j++) {     // let's do it 100 times, otherwise it's too fast!
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }
}

void Labwork::labwork1_OpenMP() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    // do something here
    
    #pragma omp parallel for
    for (int j = 0; j < 100; j++) {     // let's do it 100 times, otherwise it's too fast!
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }
}

int getSPcores(hipDeviceProp_t devProp) {
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major) {
        case 2: // Fermi
            if (devProp.minor == 1) cores = mp * 48;
            else cores = mp * 32;
            break;
        case 3: // Kepler
            cores = mp * 192;
            break;
        case 5: // Maxwell
            cores = mp * 128;
            break;
        case 6: // Pascal
            if (devProp.minor == 1) cores = mp * 128;
            else if (devProp.minor == 0) cores = mp * 64;
            else printf("Unknown device type\n");
            break;
        default:
            printf("Unknown device type\n");
            break;
    }
    return cores;
}

void Labwork::labwork2_GPU() {
    int nDevices = 0;
    // get all devices
    hipGetDeviceCount(&nDevices);
    printf("Number total of GPU : %d\n\n", nDevices);
    for (int i = 0; i < nDevices; i++){
        // get informations from individual device
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        // something more here
        printf("Device name: %s\n", prop.name);
        printf("-----\n");
        printf("Core info:\n");
        printf("Clock rate: %d\n", prop.clockRate);
        printf("Core counts: %d\n", getSPcores(prop));
        printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
        printf("Warp size: %d\n", prop.warpSize);
        printf("-----\n");
        printf("Memory info: \n");
        printf("Memory clock rate: %d\n", prop.memoryClockRate);
        printf("Bus width: %d\n", prop.memoryBusWidth);
        printf("\n");
    }

}

// rgb => gray
__global__ void grayscale(uchar3 *input, uchar3 *output) { 
    int tid = threadIdx.x + blockIdx.x * blockDim.x; 
    output[tid].x = (input[tid].x + input[tid].y +input[tid].z) / 3; 
    output[tid].z = output[tid].y = output[tid].x;
}

__global__ void grayscale_2d(uchar3 *input, uchar3 *output) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    int w = blockDim.x * gridDim.x;
    x = w * y + x;

    output[x].x = (input[x].x + input[x].y + input[x].z) / 3; 
    output[x].z = output[x].y = output[x].x;
}

void Labwork::labwork3_GPU() {
    // Calculate number of pixels
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));

    // Allocate CUDA memory  
    uchar3 *devInput;
    uchar3 *devGray;
    hipMalloc(&devInput, pixelCount * sizeof(uchar3));
    hipMalloc(&devGray, pixelCount * sizeof(uchar3));  

    // Copy CUDA Memory from CPU to GPU
    hipMemcpy(devInput, inputImage->buffer, pixelCount * 3, hipMemcpyHostToDevice);

    // Processing
    int blockSize = 64;
    int numBlock = pixelCount / blockSize;
    grayscale<<<numBlock, blockSize>>>(devInput, devGray);

    // Copy CUDA Memory from GPU to CPU
    hipMemcpy(outputImage, devGray, pixelCount * 3, hipMemcpyDeviceToHost);

    // Cleaning
    hipFree(devGray);
    hipFree(devInput);
}

void Labwork::labwork4_GPU() {
    // Calculate number of pixels
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));

    // Allocate CUDA memory  
    uchar3 *devInput;
    uchar3 *devGray;
    hipMalloc(&devInput, pixelCount * sizeof(uchar3));
    hipMalloc(&devGray, pixelCount * sizeof(uchar3));  

    // Copy CUDA Memory from CPU to GPU
    hipMemcpy(devInput, inputImage->buffer, pixelCount * 3, hipMemcpyHostToDevice);

    // Processing
    dim3 blockSize = dim3(32, 32);
    dim3 gridSize = dim3(inputImage->width / blockSize.x, inputImage->height / blockSize.y);
    grayscale_2d<<<gridSize, blockSize>>>(devInput, devGray);

    // Copy CUDA Memory from GPU to CPU
    hipMemcpy(outputImage, devGray, pixelCount * 3, hipMemcpyDeviceToHost);

    // Cleaning
    hipFree(devGray);
    hipFree(devInput);

}

void Labwork::labwork5_CPU() {
}

void Labwork::labwork5_GPU(bool shared) {
}

void Labwork::labwork6_GPU() {
}

void Labwork::labwork7_GPU() {
}

void Labwork::labwork8_GPU() {
}

void Labwork::labwork9_GPU() {

}

void Labwork::labwork10_GPU(){
}


























n